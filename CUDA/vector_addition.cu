/*
nvcc -o va vector_addition.cu
./va <N> <NUM_BLOCKS>
*/


#include <hip/hip_runtime.h>
#include <stdio.h>
#include <stdlib.h>

__global__ void add(double *a, double *b, double *c)
{
    	int index = blockIdx.x * blockDim.x + threadIdx.x;
    	c[index] = a[index] + b[index];
}

int main(int argc, char *argv[])
{
    	int N = 1048576;
    	int NUM_BLOCKS = atoi(argv[1]);

    	FILE *fp;
    	fp = fopen("vectorAddition.txt", "w");
    	double *a, *b, *c;
    	double *d_a, *d_b, *d_c;
    	double size = N * sizeof(double);

    	hipMalloc((void **)&d_a, size);
    	hipMalloc((void **)&d_b, size);
    	hipMalloc((void **)&d_c, size);

    	a = (double *)malloc(size);
    	b = (double *)malloc(size);
    	c = (double *)malloc(size);

    	for (int i = 0; i < N; i++)
    	{
        	a[i] = i;
        	b[i] = i;
    	}

    	hipMemcpy(d_a, a, size, hipMemcpyHostToDevice);
    	hipMemcpy(d_b, b, size, hipMemcpyHostToDevice);

    	add<<<NUM_BLOCKS, N / NUM_BLOCKS>>>(d_a, d_b, d_c);

    	hipMemcpy(c, d_c, size, hipMemcpyDeviceToHost);

    	for (int i = 0; i < N; i++)
        	fprintf(fp, "%f %f %f\n", a[i], b[i], c[i]);

    	free(a);
    	free(b);
    	free(c);
    	hipFree(d_a);
    	hipFree(d_b);
    	hipFree(d_c);
    	fclose(fp);
    	return 0;
}
